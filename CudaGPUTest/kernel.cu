
#include "hip/hip_runtime.h"


#include <stdio.h>
#include <cstdio>
#include <ctime>

hipError_t addWithCuda(int *c, int *a, int *b, unsigned int size);

__global__ void addKernel(int n, int *c, int *a, int *b)
{
	int index = threadIdx.x;
	int stride = blockDim.x;
	for (int i = index; i < n; i += stride)
		c[i] = a[i] + b[i];
}

int main()
{
    const int arraySize = 1 << 22;
	int *a = new int[arraySize];
	int *b = new int[arraySize];
	int *c = new int[arraySize];

	for (int i = 0; i < arraySize; i++) {
		a[i] = i;
		b[i] = i * 2;
	}

	// Variables to keep track of, to only count adding
	std::clock_t start;
	double duration;
	start = std::clock();

    // Add vectors in parallel.
    hipError_t cudaStatus = addWithCuda(c, a, b, arraySize);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addWithCuda failed!");
        return 1;
    }

    printf("{%d,%d,%d,%d,%d}\n", c[50], c[51], c[20000], c[322], c[434]);

	duration = (std::clock() - start) / (double)CLOCKS_PER_SEC * 1000.0;

	fprintf(stderr, "Time: %d ms", duration);

    // cudaDeviceReset must be called before exiting in order for profiling and
    // tracing tools such as Nsight and Visual Profiler to show complete traces.
    cudaStatus = hipDeviceReset();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceReset failed!");
        return 1;
    }

    return 0;
}

// Helper function for using CUDA to add vectors in parallel.
hipError_t addWithCuda(int *c, int *a, int *b, unsigned int size)
{
    int *dev_a = 0;
    int *dev_b = 0;
    int *dev_c = 0;

    // Allocate GPU buffers for three vectors (two input, one output)    .
    hipMalloc((void**)&dev_c, size * sizeof(int));
    hipMalloc((void**)&dev_a, size * sizeof(int));
    hipMalloc((void**)&dev_b, size * sizeof(int));

    // Copy input vectors from host memory to GPU buffers.
    hipMemcpy(dev_a, a, size * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(dev_b, b, size * sizeof(int), hipMemcpyHostToDevice);

    // Launch a kernel on the GPU with one thread for each element.
	addKernel<<<1, 256>>>(size, dev_c, dev_a, dev_b);

    // Check for any errors launching the kernel
	hipError_t cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }
    
    // cudaDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    hipDeviceSynchronize();

    // Copy output vector from GPU buffer to host memory.
    hipMemcpy(c, dev_c, size * sizeof(int), hipMemcpyDeviceToHost);

Error:
    hipFree(dev_c);
    hipFree(dev_a);
    hipFree(dev_b);
    
    return cudaStatus;
}
